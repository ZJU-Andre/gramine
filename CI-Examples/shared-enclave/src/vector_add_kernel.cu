#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA Kernel for vector addition
__global__ void vectorAddKernel(float *a, const float *b, const float *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        a[i] = b[i] + c[i];
    }
}

// C-callable wrapper function to launch the CUDA kernel
extern "C" int launch_vector_add_cuda(
    float* h_A_out, 
    const float* h_B_in, 
    const float* h_C_in, 
    int n,
    int* cuda_error_code,       // Output CUDA error code
    const char** cuda_error_str   // Output CUDA error string pointer
) {
    float *d_A = NULL, *d_B = NULL, *d_C = NULL;
    hipError_t err = hipSuccess;

    // Initialize output error pointers
    if (cuda_error_code) *cuda_error_code = hipSuccess;
    if (cuda_error_str) *cuda_error_str = hipGetErrorString(hipSuccess);

    // Allocate memory on the GPU
    err = hipMalloc((void**)&d_A, n * sizeof(float));
    if (err != hipSuccess) {
        if (cuda_error_code) *cuda_error_code = err;
        if (cuda_error_str) *cuda_error_str = hipGetErrorString(err);
        fprintf(stderr, "CUDA_WRAPPER: Failed to allocate d_A: %s\n", hipGetErrorString(err));
        goto Error;
    }

    err = hipMalloc((void**)&d_B, n * sizeof(float));
    if (err != hipSuccess) {
        if (cuda_error_code) *cuda_error_code = err;
        if (cuda_error_str) *cuda_error_str = hipGetErrorString(err);
        fprintf(stderr, "CUDA_WRAPPER: Failed to allocate d_B: %s\n", hipGetErrorString(err));
        goto Error;
    }

    err = hipMalloc((void**)&d_C, n * sizeof(float));
    if (err != hipSuccess) {
        if (cuda_error_code) *cuda_error_code = err;
        if (cuda_error_str) *cuda_error_str = hipGetErrorString(err);
        fprintf(stderr, "CUDA_WRAPPER: Failed to allocate d_C: %s\n", hipGetErrorString(err));
        goto Error;
    }

    // Copy input arrays from host to device
    err = hipMemcpy(d_B, h_B_in, n * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        if (cuda_error_code) *cuda_error_code = err;
        if (cuda_error_str) *cuda_error_str = hipGetErrorString(err);
        fprintf(stderr, "CUDA_WRAPPER: Failed to copy h_B_in to d_B: %s\n", hipGetErrorString(err));
        goto Error;
    }

    err = hipMemcpy(d_C, h_C_in, n * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        if (cuda_error_code) *cuda_error_code = err;
        if (cuda_error_str) *cuda_error_str = hipGetErrorString(err);
        fprintf(stderr, "CUDA_WRAPPER: Failed to copy h_C_in to d_C: %s\n", hipGetErrorString(err));
        goto Error;
    }

    // Define block and grid sizes
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    // Launch the kernel
    vectorAddKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, n);
    err = hipGetLastError(); // Check for errors from kernel launch
    if (err != hipSuccess) {
        if (cuda_error_code) *cuda_error_code = err;
        if (cuda_error_str) *cuda_error_str = hipGetErrorString(err);
        fprintf(stderr, "CUDA_WRAPPER: Kernel launch failed: %s\n", hipGetErrorString(err));
        goto Error;
    }

    // Synchronize device to ensure kernel completion before copying back results
    // This is good practice, though hipMemcpy DeviceToHost is blocking by default for default stream.
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        if (cuda_error_code) *cuda_error_code = err;
        if (cuda_error_str) *cuda_error_str = hipGetErrorString(err);
        fprintf(stderr, "CUDA_WRAPPER: hipDeviceSynchronize failed after kernel launch: %s\n", hipGetErrorString(err));
        goto Error;
    }

    // Copy the result array from device to host
    err = hipMemcpy(h_A_out, d_A, n * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        if (cuda_error_code) *cuda_error_code = err;
        if (cuda_error_str) *cuda_error_str = hipGetErrorString(err);
        fprintf(stderr, "CUDA_WRAPPER: Failed to copy d_A to h_A_out: %s\n", hipGetErrorString(err));
        goto Error;
    }

    // Free GPU memory
Error: // Common cleanup point
    if (d_A) hipFree(d_A);
    if (d_B) hipFree(d_B);
    if (d_C) hipFree(d_C);

    if (err != hipSuccess) {
        // If an error occurred before this point, cuda_error_code and cuda_error_str are already set.
        // If a hipFree fails, this will update them.
        hipError_t free_err = hipGetLastError(); // Check if any hipFree failed
        if (free_err != hipSuccess && err == hipSuccess) { // Only update if no prior error and free failed
             if (cuda_error_code) *cuda_error_code = free_err;
             if (cuda_error_str) *cuda_error_str = hipGetErrorString(free_err);
             fprintf(stderr, "CUDA_WRAPPER: Error during hipFree: %s\n", hipGetErrorString(free_err));
             return -1; // Indicate error from cleanup
        }
        return -1; // Indicate prior error
    }

    return 0; // Success
}
